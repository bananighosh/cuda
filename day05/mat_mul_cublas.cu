#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>
#include <hiprand.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <stdio.h>
#include <math.h>

void verify_result(float *a, float *b, float *c){
    float temp;
    float epsilon = 0.001;
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            temp = 0;
            for(int k = 0; k < n; k++){
                temp += a[k * n + i] + b[j * n + k]; //column major order
            }
            assert(fabs(c[j * n + i] - temp) < epsilon); 
        }
    }
}

int main(){
    int n = 1 << 10;
    size_t bytes = n * n * sizeof(float);

    // declare the pointers to matrices on device to host
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    // Allocate memory
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // initialize the vectors directly on GPU instead of CPU
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed
    curandSetPseudoGeneratorSeed(prng, (unsigned long long)clock());

    // Fill the matrix with random integers
    hiprandGenerateUniform(prng, d_a, n * n);
    hiprandGenerateUniform(prng, d_b, n * n)

    //cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //scaling factor
    float alpha = 1.0f;
    float beta = 0.0f;

    // calculation: (alpha * a) * b + (beta * c)
    // ( m * n ) * (n * k) = (m * k)
    // hipblasSgemm( handle, operation, operation, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc)
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, d_a, n, d_b, n, beta, d_c, n);

    //copy back all 3 matrices
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    verify_result(h_a, h_b, h_c, n);

    print("COMPLETED SUCCESSFULLY \n");

    return 0;
}
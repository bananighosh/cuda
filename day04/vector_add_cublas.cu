#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>

// Initialize a vector
void vector_init(float* a, int n){
    for(int i = 0; i < n; i++){
        a[i] = (float)(rand() % 100)
    }
}

// verify the result
void verify_result(float* a, float* b, float* c, float factor, int n){
    for(i = 0; i < n; i++){
        assert(c[i] == factor * a[i] + b[i])
    }
}

int main(){
    int n = 1 << 2;
     size_t bytes = n * sizeof(float);

     // Dec;are vector pointers
     float* h_a, h_b, h_c;
     float* d_a, d_b;

     // Allocate memory
     h_a = (float)malloc(bytes);
     h_b = (float)malloc(bytes);
     h_c = (float)malloc(bytes);
     hipMalloc(&d_a, bytes);
     hipMalloc(&d_b, bytes);

     // Initialize vectors
     vector_init(h_a, n);
     vector_init(h_b, n);

     // Create and initialize the cublas handle ( or a new context)
     hipblasHandle_t handle;
     hipblasCreate(handle);

     // Copy the vectors over to device
     hipblasSetVector(n, sizeof(float), h_a, 1, d_a, 1);
     hipblasSetVector(n, sizeof(float), h_b, 1, d_b, 1);

     //Launch a simple saxpy kernel ( single precision a * x + y)
     const float scale = 2.0f;
     hipblasSaxpy(handle, n, sizeof(float), d_a, 1, d_b, 1); // 1 represents our vectors doesnot have any empty spaces to skip

     // copy result to host
     hipblasSetVector(n, sizeof(float), d_b, 1, h_c, 1 );

     // verify 
     verify_result(h_a, h_b, h_c, scale, n);

     // clean up the cublas handle
     hipblasDestroy(handle)

     // free up memory
     hipFree(d_a);
     hipFree(d_b);
     free(h_a);
     free(h_b);

     return 0;
}